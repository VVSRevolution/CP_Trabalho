
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

# define VAL_INI 10 //valor inicial
# define LIM_SUP 200000 //limite superior (até onde vamos para calcular os numeros primos)

int main(){


    int i, j, cont;
    long long int p, q; //variaveis para armazenar dois primos (para gerar a chave publica)
    long long int n; //Multiplicação de p e q
    long long int div, numero, phi_n,e,d,tam;
    long long int *primo;
    hipMallocManaged(&primo, 100000*sizeof(long long int)); //vetor q vai aramazenar todos os numero primos

    FILE *temporPrimo;
    temporPrimo = fopen("tempoPorPrimo.txt", "a");
    

    //iniciamos nosso vetor de primos com alguns valores
    
    primo[0] = 2;
    primo[1] = 3;
    primo[2] = 5;
    primo[3] = 7;
    
    cont = 4; //contador vai ser responsavel pelos indices do nosso vetor de primos


    time_t begin = time(NULL); //função para calcular o tempo de execução


//TEST para grafico
    struct timeval stop, start;
    gettimeofday(&start, NULL);
    char int_str[20];
    sprintf(int_str, "%s", "TIMES");
    fprintf(temporPrimo, "%s\n", int_str); 


    //DESCOBRINDO NUMEROS PRIMOS
    for(numero = VAL_INI; numero < LIM_SUP; numero ++)
    {
        tam = numero/2;
        //faz a divisão do numero (n) pelos primos armazenados no vetor (primo)
        for(div = 2; div < tam; div++)
        {
            //se um dos valores primos conseguir dividir o numero ja saimos do laço.
            if(numero % div == 0)
                
                break;

            //se chegar até o fim do vetor de primos, significa q esse valor é um novo numero primo então, armazenamos ele no vetor
            if (div == tam-1){
                primo[cont] = numero;
                cont++;
                gettimeofday(&stop, NULL);
                sprintf(int_str, "%ld", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);
                fprintf(temporPrimo, "%s\n", int_str); //microsecond

            }
        }
    }
    time_t end = time(NULL); //função para calcular o tempo de execução



    printf("Quantidade de numeros primos = %d\n", cont);  //Printa a quantidade de numeros primos entre 2 e o LIM_SUP  
    printf("Tempo de execução para descobrir os numeros primos: %ld segundos\n", (end - begin)); //printa o tempo de execução em segundos para descobrir os numeros primos


    //DESCOBRINDO A CHAVE PUBLICA
    p = primo[cont-500]; // primo escolhido para ser usado na descoberta da chave primaria
    q = primo[cont-73]; // primo escolhido para ser usado na descoberta da chave primaria  

    printf("P = %lld - Q = %lld \n", p, q); 

    n = p * q;
    printf("N = %lld\n", n);

    phi_n = (p-1) * (q-1);
    printf("Phi(N) = %lld\n", phi_n);

    e = 0;
    //procura um numero primo de Phi(N) -> [numeros primos entre si são aqueles q não possuem um divisor em comum, alem do 1. Por exemplo, 25 e 9 são primos entre si]
    for(i = 10; i < phi_n; i++)
    {
        for(j = 2; j < (i + 1); j++)
        {
            if(phi_n % j == 0 && i % j == 0)
                break;

            if(j == i)
                e = i;
        }

        //E vai receber o primeiro numero primo com Phi(n) [o algoritimo em si é um numero aleatorio mas o primeiro serve]
        if(e != 0)
            break;
    }

    printf("E = %lld\n", e);

    printf("CHAVE PUBLICA: [%lld, %lld]\n", n, e);
    


    time_t begin_2 = time(NULL); //função para calcular o tempo de execução
    for (d = 1; d <= phi_n; d++)
    {
        if((d*e) % phi_n == 1)
            break;
    }
    time_t end_2 = time(NULL); //função para calcular o tempo de execução

    printf("Tempo de execução para calcular a chave privada: %ld segundos\n", (end_2 - begin_2)); //printa o tempo de execução em segundos para descobrir os numeros primos

    printf("CHAVE PRIVADA: [%lld]\n", d);
    
    hipFree(primo);
    fclose(temporPrimo);
    return 0;
}